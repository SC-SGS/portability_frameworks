
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

// the CUDA compute kernel
__global__ void saxpy(const double alpha, const double *X, double *Y, const int N) {
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        Y[idx] = alpha * X[idx] + Y[idx];
    }
}

int main() {
    const int N = 1024;

    // create and fill used data
    std::vector<double> X(N);
    std::vector<double> Y(N);
    for (int i = 0; i < N; ++i) {
        X[i] = i;
        Y[i] = 2 * i;
    }
    const double alpha = 0.5;

    double *d_X, *d_Y;
    // allocate memory on the device
    hipMalloc(&d_X, N * sizeof(double));
    hipMalloc(&d_Y, N * sizeof(double));
    // copy data to the device
    hipMemcpy(d_X, X.data(), N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Y, Y.data(), N * sizeof(double), hipMemcpyHostToDevice);

    // perform computations
    saxpy<<<(N + 255) / 256, 256>>>(alpha, d_X, d_Y, N);

    // copy data to the host
    hipMemcpy(Y.data(), d_Y, N * sizeof(double), hipMemcpyDeviceToHost);
    // free the resources
    hipFree(d_X);
    hipFree(d_Y);

    for (int i = 0; i < 10; ++i) {
        std::cout << Y[i] << ' ';
    }
    std::cout << std::endl;

    return 0;
}
